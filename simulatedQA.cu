#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <random>

#define K 0.5

texture<float, 1, hipReadModeElementType> Q_text;
texture<int, 1, hipReadModeElementType> s_text;
texture<int, 1, hipReadModeElementType> pre_s_text;

void cudaErr(hipError_t err)
{
    if (err != hipSuccess)
    {
        printf("%s, %s\n", hipGetErrorName(err), hipGetErrorString(err));
        exit(1);
    }
}

float energy(int *s, float *Q, int dim)
{
    double *temp;
    temp = (double *)malloc(sizeof(double) * dim);
    float sum = 0;
    for (int i = 0; i < dim; i++)
    {
        for (int j = 0; j < dim; j++)
        {
            temp[i] += s[j] * Q[i * dim + j];
        }
    }
    for (int i = 0; i < dim; i++)
    {
        sum += temp[i] * s[i];
    }
    free(temp);
    return sum;
}

__global__ void calculate(int *trotterBlock, int spinIdx, int dim, int trotterNum, int beta, int seed)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    // if (index < dim)
    if (index == 0)
    {
        hiprandState state;
        hiprand_init(seed, index, 0, &state);

        // calculate<<<1, 1>>>(trotterBlock[j * dim + i - j]);
        int quboRowIdx = spinIdx % dim;
        float sum = 0;
        for (int i = 0; i < dim; i++)
        {
            sum += tex1Dfetch(s_text, spinIdx) * tex1Dfetch(Q_text, quboRowIdx * dim + i);
        }

        // calcPre<<<1, 1>>>(trotterBlock[(j - 1) * dim + i - j]);
        if (spinIdx - dim >= 0)
        {
            // sum -= tex1Dfetch(s_text, spinIdx - dim) * K;
        }
        else
        {
            // sum -= K;
        }

        // calcAfter<<<1, 1>>>(prevTrotterBlock[(j + 1) * dim + i - j]);
        if (spinIdx + dim < dim * trotterNum)
        {
            // sum += tex1Dfetch(pre_s_text, spinIdx + dim) * K;
        }
        else
        {
            // sum += K;
        }

        // check flip
        if (exp(-1 * sum / beta) > hiprand_uniform(&state))
        {
            trotterBlock[spinIdx] *= -1;
        }
    }
}

extern "C"
{
    float simulatedQA(int *s, float *Q, int dim, int trotterNum, int totalSweeps);
}

float simulatedQA(int *s, float *Q, int dim, int trotterNum, int totalSweeps)
{
    srand(1);

    int *trotterBlock;
    hipMalloc(&trotterBlock, trotterNum * dim * sizeof(int));

    int *trotterBlockLocal;
    hipHostMalloc(&trotterBlockLocal, trotterNum * dim * sizeof(int));

    int *prevTrotterBlock;
    hipMalloc(&prevTrotterBlock, trotterNum * dim * sizeof(int));

    for (int i = 0; i < trotterNum; i++)
    {
        hipMemcpy(&trotterBlockLocal[i * dim], s, dim * sizeof(int), hipMemcpyHostToHost);
    }

    hipMemcpy(trotterBlock, trotterBlockLocal, dim * trotterNum * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(prevTrotterBlock, trotterBlock, dim * trotterNum * sizeof(int), hipMemcpyDeviceToDevice);

    float *Q_dev;
    hipMalloc(&Q_dev, dim * dim * sizeof(float));
    cudaErr(hipMemcpy(Q_dev, Q, dim * dim * sizeof(float), hipMemcpyHostToDevice));

    cudaErr(hipBindTexture(0, Q_text, Q_dev, dim * dim * sizeof(float)));
    cudaErr(hipBindTexture(0, s_text, trotterBlock, dim * trotterNum * sizeof(int)));
    cudaErr(hipBindTexture(0, pre_s_text, prevTrotterBlock, dim * trotterNum * sizeof(int)));

    for (int sweep = 1; sweep <= totalSweeps; sweep++)
    {
        for (int i = 0; i < dim + trotterNum - 1; i++)
        {
            for (int j = 0; j <= i && j < trotterNum; j++)
            {
                if (i - j < dim)
                {
                    hipStream_t stream;
                    hipStreamCreate(&stream);
                    // calculate in parallel psudocode //
                    calculate<<<32, 32, 0, stream>>>(trotterBlock, j * dim + i - j, dim, trotterNum, sweep, rand());
                    hipStreamDestroy(stream);
                }
            }
            hipDeviceSynchronize();
        }
        hipMemcpy(prevTrotterBlock, trotterBlock, trotterNum * dim * sizeof(int), hipMemcpyDeviceToDevice);

        hipMemcpy(trotterBlockLocal, trotterBlock, trotterNum * dim * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(s, &trotterBlockLocal[dim * (trotterNum - 1)], dim * sizeof(int), hipMemcpyHostToHost);
        printf("%.9f\n", energy(s, Q, dim));
    }

    hipMemcpy(trotterBlockLocal, trotterBlock, trotterNum * dim * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(s, &trotterBlockLocal[dim * (trotterNum - 1)], dim * sizeof(int), hipMemcpyHostToHost);

    hipUnbindTexture(Q_text);
    hipUnbindTexture(s_text);
    hipUnbindTexture(pre_s_text);

    hipFree(trotterBlock);
    hipFree(prevTrotterBlock);
    hipFree(Q_dev);
    hipHostFree(trotterBlockLocal);

    return energy(s, Q, dim);
}