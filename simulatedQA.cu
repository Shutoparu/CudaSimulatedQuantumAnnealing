#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <random>

#define K 0.5

texture<float, 1, hipReadModeElementType> Q_text;
texture<int, 1, hipReadModeElementType> s_text;
texture<int, 1, hipReadModeElementType> pre_s_text;

/**
 * @brief used to check cuda error
 *
 * @param err cuda error to be checked
 */
void cudaErr (hipError_t err) {
    if (err != hipSuccess) {
        printf ("%s, %s\n", hipGetErrorName (err), hipGetErrorString (err));
        exit (1);
    }
}

/**
 * @brief function to calculate energy. currently running on cpu
 *
 * @param s binary array
 * @param Q qubo matrix
 * @param dim dimention of the array
 * @return calculated energy.
 */
float energy (int* s, float* Q, int dim) {
    float* temp;
    temp = (float*)malloc (sizeof (float) * dim);
    float sum = 0;
    for (int i = 0; i < dim; i++) {
        for (int j = 0; j < dim; j++) {
            temp[i] += s[j] * Q[i * dim + j];
        }
    }
    for (int i = 0; i < dim; i++) {
        sum += temp[i] * s[i];
        temp[i] = 0;
    }
    free (temp);
    return sum;
}

/**
 * @brief create the beta array
 *
 * @param betaStart starting value of beta
 * @param betaStop ending value of beta
 * @param beta the beta array to be returned
 * @param sweeps the length of beta array
 */
void getAnnealingBeta (int betaStart, int betaStop, float* beta, int sweeps) {

    float logBetaStart = log ((float)betaStart);
    float logBetaStop = log ((float)betaStop);
    float logBetaRange = (logBetaStop - logBetaStart) / (float)sweeps;
    for (int i = 0; i < sweeps; i++) {
        beta[i] = exp (logBetaStart + logBetaRange * i);
    }
}

/**
 * @brief used to determine whether to flip a bit or not
 *
 * @param trotterBlock all trotters in an array
 * @param spinIdx the index of the checked bit in the trotter block
 * @param dim dimetnion of the bit array
 * @param trotterNum numbers of trotters
 * @param beta used to determine passing threshold
 * @param seed used to generate random number
 */
__global__ void calculate (int* trotterBlock, int spinIdx, int dim, int trotterNum, int beta, int seed) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    // if (index < dim)
    if (index == 0) {
        hiprandState state;
        hiprand_init (seed, index, 0, &state);

        // calculate<<<1, 1>>>(trotterBlock[j * dim + i - j]);
        int quboRowIdx = spinIdx % dim;
        float delta_E = 0;
        for (int i = 0; i < dim; i++) {
            delta_E += tex1Dfetch (s_text, (spinIdx / dim) * dim + i) * tex1Dfetch (Q_text, quboRowIdx * dim + i);
        }

        // calcPre<<<1, 1>>>(trotterBlock[(j - 1) * dim + i - j]);
        if (spinIdx - dim >= 0) {
            delta_E -= tex1Dfetch (s_text, spinIdx - dim) * K;
        } else {
            delta_E -= tex1Dfetch (pre_s_text, dim * (trotterNum - 1) + quboRowIdx) * K;
        }

        // calcAfter<<<1, 1>>>(prevTrotterBlock[(j + 1) * dim + i - j]);
        if (spinIdx + dim < dim * trotterNum) {
            delta_E += tex1Dfetch (pre_s_text, spinIdx + dim) * K;
        } else {
            delta_E += tex1Dfetch (pre_s_text, quboRowIdx) * K;
        }

        // check flip
        if (exp (-1 * delta_E / beta) > 1) {
            trotterBlock[spinIdx] *= -1;
        }
    }
}

extern "C"
{
    float simulatedQA (int* s, float* Q, int dim, int trotterNum, int totalSweeps);
}

/**
 * @brief the code to run a simulated quantum annealing algorithm
 *
 * @param s binary array
 * @param Q qubo matrix
 * @param dim size of binary array
 * @param trotterNum numbers of trotters needed
 * @param totalSweeps numbers of monte carlo steps
 * @return the final energy after the algorithm
 */
float simulatedQA (int* s, float* Q, int dim, int trotterNum, int totalSweeps) {
    srand (1);

    int* trotterBlock;
    hipMalloc (&trotterBlock, trotterNum * dim * sizeof (int));

    int* trotterBlockLocal;
    hipHostMalloc (&trotterBlockLocal, trotterNum * dim * sizeof (int));

    int* prevTrotterBlock;
    hipMalloc (&prevTrotterBlock, trotterNum * dim * sizeof (int));

    for (int i = 0; i < trotterNum * dim; i++) {
        trotterBlockLocal[i] = (int)(((int)(rand () / RAND_MAX) - 0.5) * 2);
    }

    cudaErr (hipMemcpy (trotterBlock, trotterBlockLocal, dim * trotterNum * sizeof (int), hipMemcpyHostToDevice));
    cudaErr (hipMemcpy (prevTrotterBlock, trotterBlock, dim * trotterNum * sizeof (int), hipMemcpyDeviceToDevice));

    float* beta;
    hipMallocManaged (&beta, totalSweeps * sizeof (float));

    float betaStart = 1;
    float betaEnd = 100;

    getAnnealingBeta (betaStart, betaEnd, beta, totalSweeps);

    float* Q_dev;
    cudaErr (hipMalloc (&Q_dev, dim * dim * sizeof (float)));
    cudaErr (hipMemcpy (Q_dev, Q, dim * dim * sizeof (float), hipMemcpyHostToDevice));

    cudaErr (hipBindTexture (0, Q_text, Q_dev, dim * dim * sizeof (float)));
    cudaErr (hipBindTexture (0, s_text, trotterBlock, dim * trotterNum * sizeof (int)));
    cudaErr (hipBindTexture (0, pre_s_text, prevTrotterBlock, dim * trotterNum * sizeof (int)));

    for (int sweep = 0; sweep < totalSweeps; sweep++) {
        for (int i = 0; i < dim + trotterNum - 1; i++) {
            for (int j = 0; j < trotterNum && j <= i; j++) {
                if (i - j < dim) {
                    hipStream_t stream;
                    hipStreamCreate (&stream);
                    // calculate in parallel psudocode //
                    calculate << <32, 32, 0, stream >> > (trotterBlock, j * dim + i - j, dim, trotterNum, beta[sweep], rand ());
                    hipStreamDestroy (stream);
                }
            }
            hipDeviceSynchronize ();
        }
        hipMemcpy (prevTrotterBlock, trotterBlock, trotterNum * dim * sizeof (int), hipMemcpyDeviceToDevice);

        hipMemcpy (trotterBlockLocal, trotterBlock, trotterNum * dim * sizeof (int), hipMemcpyDeviceToHost);
        hipMemcpy (s, &trotterBlockLocal[dim * (trotterNum - 1)], dim * sizeof (int), hipMemcpyHostToHost);
        for (int i = 0; i < dim; i++) {
            printf ("%2d ", s[i]);
        }
        printf ("\n");
        // printf("%.9f\n", energy(s, Q, dim));
    }

    float en;

    hipMemcpy (trotterBlockLocal, trotterBlock, trotterNum * dim * sizeof (int), hipMemcpyDeviceToHost);
    hipMemcpy (s, &trotterBlockLocal[dim * (trotterNum - 1)], dim * sizeof (int), hipMemcpyHostToHost);

    hipUnbindTexture (Q_text);
    hipUnbindTexture (s_text);
    hipUnbindTexture (pre_s_text);

    hipFree (trotterBlock);
    hipFree (prevTrotterBlock);
    hipFree (Q_dev);
    hipHostFree (trotterBlockLocal);

    en = energy (s, Q, dim);
    printf ("%f\n", en);
    return en;
}