#include "hip/hip_runtime.h"
#include <stdio.h>

texture<int, 1, hipReadModeElementType> text;

__global__ void printTexture(int num, int* out){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx<num){
        out[idx] = tex1Dfetch(text,idx);
    }
}

int main(){

    int * local;
    hipHostMalloc(&local, 5*sizeof(int));
    for(int i=0; i<5; i++){
        local[i] = i;
    }
    
    int* device;
    hipMalloc(&device, 5*sizeof(int));
    hipMemcpy(device, local, 5*sizeof(int), hipMemcpyHostToDevice);

    hipBindTexture(0, text, device, 5*sizeof(int));

    int *out;
    hipMallocManaged(&out, 5*sizeof(int));

    printTexture<<<3,3>>>(5, out);
    hipDeviceSynchronize();

    for(int i=0; i<5; i++){
        printf("%d\n",out[i]);
    }

}